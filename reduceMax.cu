#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define SEED 31415926

#define GTX1080Ti 1
#define GTX1070 2
#define GPU GTX1080Ti

#if GPU == GTX1080Ti
#define MP 28
#define THREADS_PER_BLOCK 1024
#define RESIDENT_BLOCKS_PER_MP 2
#endif

#if GPU == GTX1070
#define MP 28
#define THREADS_PER_BLOCK 1024
#define RESIDENT_BLOCKS_PER_MP 2
#endif

#define TOTAL_BLOCKS (MP * RESIDENT_BLOCKS_PER_MP)
#define NTA (TOTAL_BLOCKS * THREADS_PER_BLOCK)

__device__ float blockMax[TOTAL_BLOCKS];

__global__ void reduceMax_persist(float *max, float *Input,
                                  unsigned int nElements) {
  // Inicia vetor em shared memory com máximo de cada thread
  __shared__ float threadsMax[THREADS_PER_BLOCK];
  int t = threadIdx.x;
  threadsMax[t] = 0;

  // FASE 1 - Computa o máximo para cada thread
  int initial = blockDim.x * blockIdx.x + t;
  for (int i = initial; i < nElements; i += NTA)
    threadsMax[t] = fmax(threadsMax[t], Input[i]);

  // FASE 2 - Computa o máximo do bloco usando o algoritmo dos slides
  for (int stride = blockDim.x; stride > 0; stride /= 2) {
    __syncthreads();
    if (t < stride)
      threadsMax[t] = fmax(threadsMax[t], threadsMax[t + stride]);
  }

  // FASE 3 - Computa o máximo de todos os blocos usando atomicos
  int b = blockIdx.x;
  if (t == 0) {
    blockMax[b] = threadsMax[0];
    for (int stride = TOTAL_BLOCKS; stride > 0; stride /= 2) {
      __syncthreads();
      if (b < stride)
        blockMax[b] = fmax(blockMax[b], blockMax[b + stride]);
    }
  }

  if (b == 0 && t == 0)
    *max = blockMax[0];
}

__global__ void reduceMax_atomic_persist(float *max, float *Input,
                                         unsigned int nElements) {
  return;
}

int main(int argc, char **argv) {
  if (argc != 2) {
    fprintf(stderr, "Uso: %s <TAM_VETOR>\n", argv[0]);
    exit(EXIT_FAILURE);
  }

  srand(SEED);

  int vectorSize = atoi(argv[1]);

  // Aloca espaço no host para vetor A e resultado
  float *h_A = (float *)malloc(vectorSize * sizeof(float));
  float h_max = 0;

  // Aloca espaço na GPU para vetor A e resultado
  float *d_A = NULL;
  float *d_max = NULL;
  hipMalloc((void **)&d_A, vectorSize * sizeof(float));
  hipMalloc((void **)&d_max, sizeof(float));

  // Inicializa vetor A com valores aleatórios entre 0 e 1
  for (int i = 0; i < vectorSize; i++)
    h_A[i] = (float)rand() / RAND_MAX;

  // Copia vetor A para GPU
  hipMemcpy(h_A, d_A, vectorSize * sizeof(float), hipMemcpyHostToDevice);

  // Lança kernel
  reduceMax_persist<<<TOTAL_BLOCKS, THREADS_PER_BLOCK>>>(d_max, d_A, vectorSize);

  // Copia resultado para o host
  hipMemcpy(&h_max, d_max, sizeof(float), hipMemcpyDeviceToHost);

  // Calcula redução normal em CPU
  float correct = 0;
  for (int i = 0; i < vectorSize; i++)
    correct = fmax(h_A[i], correct);

  // Checa corretude do resultado
  if (h_max != correct) {
    fprintf(stderr, "Resultado errado. Esperava %f e obteve %f\n", correct,
            h_max);
    exit(EXIT_FAILURE);
  }

  // Libera estruturas
  hipFree(d_A);
  hipFree(d_max);
  free(h_A);
}
