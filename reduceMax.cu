#include "hip/hip_runtime.h"
#include <math.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define SEED 31415926

#define GTX1080Ti 1
#define GTX1070 2
#define GPU GTX1080Ti

#if GPU == GTX1080Ti
#define MP 28
#define THREADS_PER_BLOCK 1024
#define RESIDENT_BLOCKS_PER_MP 2
#endif

#if GPU == GTX1070
#define MP 28
#define THREADS_PER_BLOCK 1024
#define RESIDENT_BLOCKS_PER_MP 2
#endif

#define TOTAL_BLOCKS (MP * RESIDENT_BLOCKS_PER_MP)
#define NTA (TOTAL_BLOCKS * THREADS_PER_BLOCK)

__device__ float blockMax[TOTAL_BLOCKS];

__global__ void reduceMax_persist(float *max, float *Input,
                                  unsigned int nElements) {
  // Inicia vetor em shared memory com máximo de cada thread
  __shared__ float threadsMax[THREADS_PER_BLOCK];
  int t = threadIdx.x;
  threadsMax[t] = 0;

  // FASE 1 - Computa o máximo para cada thread
  int initial = blockDim.x * blockIdx.x + t;
  for (int i = initial; i < nElements; i += NTA)
    threadsMax[t] = fmaxf(threadsMax[t], Input[i]);

  // FASE 2 - Computa o máximo do bloco usando o algoritmo dos slides
  for (int stride = blockDim.x; stride > 0; stride /= 2) {
    __syncthreads();
    if (t < stride && t + stride < THREADS_PER_BLOCK)
      threadsMax[t] = fmaxf(threadsMax[t], threadsMax[t + stride]);
  }

  // FASE 3 - Computa o máximo de todos os blocos usando atomicos
  int b = blockIdx.x;
  if (t == 0) {
    blockMax[b] = threadsMax[0];
    for (int stride = TOTAL_BLOCKS; stride > 0; stride /= 2) {
      __syncthreads();
      if (b < stride && b + stride < TOTAL_BLOCKS)
        blockMax[b] = fmaxf(blockMax[b], blockMax[b + stride]);
    }
  }

  if (b == 0 && t == 0)
    *max = blockMax[0];
}

__global__ void reduceMax_atomic_persist(float *max, float *Input,
                                         unsigned int nElements) {
  return;
}

void errorAndAbort(const char *format, ...) {
  va_list args;
  va_start(args, format);
  vfprintf(stderr, format, args);
  va_end(args);

  printf("Abortado\n");
  exit(EXIT_FAILURE);
}

int main(int argc, char **argv) {
  hipError_t err = hipSuccess;

  if (argc != 2)
    errorAndAbort("Uso: %s <TAM_VETOR>\n", argv[0]);

  srand(SEED);

  int vectorSize = atoi(argv[1]);

  // Aloca espaço no host para vetor A e resultado
  float *h_A = (float *)malloc(vectorSize * sizeof(float));
  float h_max = 0;

  // Aloca espaço na GPU para vetor A e resultado
  float *d_A = NULL;
  float *d_max = NULL;
  err = hipMalloc((void **)&d_A, vectorSize * sizeof(float));
  if (err != hipSuccess)
    errorAndAbort("Erro ao alocar vetor A no dispositivo: %s\n",
                  hipGetErrorString(err));
  err = hipMalloc((void **)&d_max, sizeof(float));
  if (err != hipSuccess)
    errorAndAbort("Erro ao alocar resultado no dispositivo: %s\n",
                  hipGetErrorString(err));

  // Inicializa vetor A com valores aleatórios
  for (int i = 0; i < vectorSize; i++) {
    float a = rand();
    float b = rand();
    h_A[i] = a * 100 + b;
  }

  // Copia vetor A para GPU
  err =
      hipMemcpy(d_A, h_A, vectorSize * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess)
    errorAndAbort("Erro ao copiar vetor A para dispositivo: %s\n",
                  hipGetErrorString(err));

  // Lança kernel
  reduceMax_persist<<<TOTAL_BLOCKS, THREADS_PER_BLOCK>>>(d_max, d_A,
                                                         vectorSize);
  err = hipGetLastError();
  if (err != hipSuccess)
    errorAndAbort("Erro ao lançar kernel reduceMax_persist: %s\n",
                  hipGetErrorString(err));

  // Copia resultado para o host
  err = hipMemcpy(&h_max, d_max, sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess)
    errorAndAbort("Erro ao copiar resultado para o host: %s\n",
                  hipGetErrorString(err));

  // Calcula redução normal em CPU
  float correct = 0;
  for (int i = 0; i < vectorSize; i++)
    correct = fmaxf(h_A[i], correct);

  // Checa corretude do resultado
  if (fabsf(h_max - correct) > 1e5)
    errorAndAbort("Resultado errado. Esperava %f e obteve %f\n", correct,
                  h_max);

  // Libera estruturas
  hipFree(d_A);
  hipFree(d_max);
  free(h_A);
}
