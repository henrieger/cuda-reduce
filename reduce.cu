#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define SEED 31415926

#define GTX1080Ti 1
#define GTX1070 2
#define GPU GTX1080Ti

#if GPU == GTX1080Ti
    #define MP 28
    #define THREADS_PER_BLOCK 1024
    #define RESIDENT_BLOCKS_PER_MP 2
    #define NTA (MP*RESIDENT_BLOCKS_PER_MP*THREADS_PER_BLOCK)
#endif

#if GPU == GTX1070
    #define MP 28
    #define THREADS_PER_BLOCK 1024
    #define RESIDENT_BLOCKS_PER_MP 2
    #define NTA (MP*RESIDENT_BLOCKS_PER_MP*THREADS_PER_BLOCK)
#endif 

__global__ void reduce(const float *A, float *r) {
  return;
}

int main(int argc, char **argv) {
  if (argc != 2) {
    fprintf(stderr, "Uso: %s <TAM_VETOR>", argv[0]);
    exit(EXIT_FAILURE);
  }

  srand(SEED);

  int vectorSize = atoi(argv[1]);

  // Aloca espaço no host para vetor A e resultado
  float *h_A = (float *)malloc(vectorSize * sizeof(float));
  float h_result = 0;

  // Aloca espaço na GPU para vetor A e resultado
  float *d_A = NULL;
  float *d_result = NULL;
  hipMalloc((void **)&d_A, vectorSize * sizeof(float));
  hipMalloc((void **)&d_result, sizeof(float));

  // Inicializa vetor A com valores aleatórios entre 0 e 1
  for (int i = 0; i < vectorSize; i++)
    h_A[i] = (float)rand()/RAND_MAX;

  // Copia vetor A para GPU
  hipMemcpy(h_A, d_A, vectorSize * sizeof(float), hipMemcpyHostToDevice);

  // Lança kernel
  // TODO

  // Copia resultado para o host
  hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);


  // Calcula redução normal em CPU
  float correct = 0;
  for (int i = 0; i < vectorSize; i++)
    correct = fmax(h_A[i], correct);

  // Checa corretude do resultado
  if (h_result != correct) {
    fprintf(stderr, "Resultado errado. Esperava %f e obteve %f", correct, h_result);
    exit(EXIT_FAILURE);
  }

  // Libera estruturas
  hipFree(d_A);
  hipFree(d_result);
  free(h_A);
}
